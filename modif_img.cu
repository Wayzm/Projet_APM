#include <iostream>
#include <string.h>
#include <stdlib.h>
#include "FreeImage.h"

#define WIDTH 1920 // I genuinely don't know why these values exist
#define HEIGTH 1024
#define BPP 24
#define ui32 unsigned int

using namespace std;


int main(int argc, char** argv){

    FreeImage_Initialise();
    const char* PathName="img.jpg";
    const char* PathDest="new_img.png";

    ui32* d_img;

    // load and decode a regular file
    FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
    if(fif == FIF_UNKNOWN){
        perror("The image either has no signature or the recognition is failing.\n");
        exit(1);
    }
    FIBITMAP* bitmap = FreeImage_Load(fif, PathName, 0);
    if(!bitmap){
        perror("Failed image memory allocation.\n");
        exit(1);
    }

    const ui32 width = FreeImage_GetWidth(bitmap);
    const ui32 height = FreeImage_GetHeight(bitmap);
    const ui32 pitch = FreeImage_GetPitch(bitmap);
    const ui32 IMG_SIZE = sizeof(ui32) * width * height;
    fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

    // Array of IMG
    ui32* img = (ui32*)malloc(sizeof(ui32) * 3 * IMG_SIZE);
    if(img == NULL){
        perror("Memory allocation for img array failed.\n");
        exit(1);
    }
    ui32* h_img = (ui32*)malloc(sizeof(ui32) * 3 * IMG_SIZE);
    if(h_img == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }

    // RED, BLUE and GREEN pixels of IMG on device
    hipMalloc((void**)&d_img, 3 * IMG_SIZE);

    BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
    for (ui32 y = 0U; y < height; ++y){
      BYTE *pixel = (BYTE*)bits;
      for (ui32 x = 0U; x < width; ++x){
        int idx = ((y * width) + x) * 3;
        img[idx + 0] = pixel[FI_RGBA_RED];
        img[idx + 1] = pixel[FI_RGBA_GREEN];
        img[idx + 2] = pixel[FI_RGBA_BLUE];
        pixel += 3;
      }
      // next line
      bits += pitch;
    }

    free(img);
    free(h_img);
    return 0;
}