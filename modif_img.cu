#include <iostream>
#include <string.h>
#include <stdlib.h>
#include "FreeImage.h"

#define WIDTH 1920 // I genuinely don't know why these values exist
#define HEIGTH 1024
#define BPP 24
#define ui32 unsigned int

using namespace std;


int main(int argc, char** argv){

    FreeImage_Initialise();
    const char* PathName="img.jpg";
    const char* PathDest="new_img.png";

    ui32* dr_img, db_img, dg_img, hr_img, hb_img, hg_img;

    // load and decode a regular file
    FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
    if(fif == FIF_UNKNOWN){
        perror("The image either has no signature or the recognition is failing.\n");
        exit(1);
    }
    FIBITMAP* bitmap = FreeImage_Load(fif, PathName, 0);
    if(!bitmap){
        perror("Failed image memory allocation.\n");
        exit(1);
    }

    const ui32 width = FreeImage_GetWidth(bitmap);
    const ui32 height = FreeImage_GetHeight(bitmap);
    const ui32 pitch = FreeImage_GetPitch(bitmap);
    const ui32 IMG_SIZE = width * height;
    fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

    // Array of IMG
    ui32* img = (ui32*)malloc(sizeof(ui32) * 3 * IMG_SIZE);
    if(img == NULL){
        perror("Memory allocation for img array failed.\n");
        exit(1);
    }
    ui32* h_tmp = (ui32*)malloc(sizeof(ui32) * 3 * IMG_SIZE);
    if(h_tmp == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    // RED, BLUE and GREEN pixels of IMG on host
    hipHostMalloc((void**)&hr_img, IMG_SIZE);
    hipHostMalloc((void**)&hb_img, IMG_SIZE);
    hipHostMalloc((void**)&hg_img, IMG_SIZE);
    // RED, BLUE and GREEN pixels of IMG on device
    hipMalloc((void**)&dr_img, IMG_SIZE);
    hipMalloc((void**)&db_img, IMG_SIZE);
    hipMalloc((void**)&dg_img, IMG_SIZE);

    BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
    for (ui32 y = 0U; y < height; ++y){
      BYTE *pixel = (BYTE*)bits;
      for (ui32 x = 0U; x < width; ++x){
        int idx = ((y * width) + x) * 3;
        img[idx + 0] = pixel[FI_RGBA_RED];
        img[idx + 1] = pixel[FI_RGBA_GREEN];
        img[idx + 2] = pixel[FI_RGBA_BLUE];
        pixel += 3;
      }
      // next line
      bits += pitch;
    }

    free(img);
    return 0;
}