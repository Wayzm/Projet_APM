#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <cstdlib>
#include <cstdio>
#include <stdlib.h>
#include "FreeImage.h"

#define WIDTH 3840
#define HEIGHT 2160
#define BPP 24
#define ui32 unsigned int

using namespace std;

//commentaire
__global__ void horizontal_sym(ui32* d_img, ui32* d_tmp, ui32 width, ui32 height){

    // Compute thread id
    const ui32 x = threadIdx.x + blockDim.x * blockIdx.x;
    const ui32 y = threadIdx.y + blockDim.y * blockIdx.y;
    const ui32 idx = y * (3 * width) + x;
    // Compute target destination
    const ui32 xT = x;
    const ui32 yT = height - y;
    const ui32 idxT = yT * 3 * width + xT;
    // Flipping the image
    d_tmp[idxT] = d_img[idx];
}

__global__ void grey_img(ui32* dr, ui32* dg, ui32* db, ui32 width, ui32 height){

    // Compute thread id
    const ui32 x = threadIdx.x + blockDim.x * blockIdx.x;
    const ui32 y = threadIdx.y + blockDim.y * blockIdx.y;
    const ui32 idx = y * (width) + x;
    // Compute greyed pixel
    float value = dr[idx] * 0.299 + dg[idx] * 0.587 + db[idx] * 0.114;
    // Copy back the greyed pixel
    dr[idx] = value;
    dg[idx] = value;
    db[idx] = value;
}

/*** DO NOT FORGET TO GREY SCALE FIRST***/
__global__ void sobel(){
    ui32 value[3][3];
    // Compute thread id
    const ui32 x = threadIdx.x + blockDim.x * blockIdx.x;
    const ui32 y = threadIdx.y + blockDim.y * blockIdx.y;
    // const ui32 idx = y * (width) + x;
    /*** SOBEL ALGORITHM ***/
    /**  Sobel matrix x threads of each block
    ** | -1 0 1 |   | (0 0) (0 1) (0 2) |
    ** | -2 0 2 | x | (1 0) (1 1) (1 2) |
    ** | -1 0 1 |   | (2 0) (2 0) (2 2) |
    **/

    // Copy back the greyed pixel
    __syncthreads();
    // dr[idx] = value;
    // dg[idx] = value;
    // db[idx] = value;
}

int main(int argc, char** argv){

    FreeImage_Initialise();
    const char* PathName="img.jpg";
    const char* PathDest="new_img.png";

    ui32* d_img;
    ui32* d_tmp;
    ui32* dr;
    ui32* dg;
    ui32* db;

    // load and decode a regular file
    FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
    if(fif == FIF_UNKNOWN){
        perror("The image either has no signature or the recognition is failing.\n");
        exit(1);
    }
    FIBITMAP* bitmap = FreeImage_Load(fif, PathName, 0);
    if(!bitmap){
        perror("Failed image memory allocation.\n");
        exit(1);
    }

    const ui32 width = FreeImage_GetWidth(bitmap);
    const ui32 height = FreeImage_GetHeight(bitmap);
    const ui32 pitch = FreeImage_GetPitch(bitmap);
    const ui32 IMG_SIZE = sizeof(ui32) * width * height;
    fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

    // Array of IMG
    ui32* img = (ui32*)malloc(sizeof(ui32) * 3 * IMG_SIZE);
    if(img == NULL){
        perror("Memory allocation for img array failed.\n");
        exit(1);
    }

    ui32* hr = (ui32*)malloc(IMG_SIZE);
    if(hr == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    ui32* hg = (ui32*)malloc(IMG_SIZE);
    if(hg == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    ui32* hb = (ui32*)malloc(IMG_SIZE);
    if(hb == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    // RED, BLUE and GREEN pixels of IMG on device
    hipMalloc((void**)&d_img, 3 * IMG_SIZE);
    hipMalloc((void**)&d_tmp, 3 * IMG_SIZE);
    hipMalloc((void**)&dr, IMG_SIZE);
    hipMalloc((void**)&dg, IMG_SIZE);
    hipMalloc((void**)&db, IMG_SIZE);

    BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
    for (ui32 y = 0U; y < height; ++y){
      BYTE *pixel = (BYTE*)bits;
      for (ui32 x = 0U; x < width; ++x){
        int idx = ((y * width) + x) * 3;
        img[idx + 0] = pixel[FI_RGBA_RED];
        img[idx + 1] = pixel[FI_RGBA_GREEN];
        img[idx + 2] = pixel[FI_RGBA_BLUE];
        pixel += 3;
      }
      // next line
      bits += pitch;
    }

    for(ui32 y = 0U; y < height; ++y){
        for(ui32 x = 0U; x < 3 * width; ++x){
            int idx = ((y * width) + x) * 3;
            hr[y * width + x] = img[idx + 0];
            hg[y * width + x] = img[idx + 1];
            hb[y * width + x] = img[idx + 2];
        }
    }

    // 1024 threads per blocs in 2D
    dim3 Threads_Per_Blocks(32, 32);
    // Blocks for inverted image
    dim3 Num_Blocks(3 * width/Threads_Per_Blocks.x, height/Threads_Per_Blocks.y);
    // Blocks for gray scale
    // dim3 Num_Blocks(width/Threads_Per_Blocks.x + 1, height/Threads_Per_Blocks.y + 1);

    // Copy to device
    hipMemcpy(d_img, img, 3 * IMG_SIZE, hipMemcpyHostToDevice);
    // hipMemcpy(dr, hr, IMG_SIZE, hipMemcpyHostToDevice);
    // hipMemcpy(dg, hg, IMG_SIZE, hipMemcpyHostToDevice);
    // hipMemcpy(db, hb, IMG_SIZE, hipMemcpyHostToDevice);

    //Horizontal Symetry
    horizontal_sym<<<Num_Blocks, Threads_Per_Blocks>>>(d_img, d_tmp, width, height);
    // Copy to Host
    hipMemcpy(img, d_tmp, 3 * IMG_SIZE, hipMemcpyDeviceToHost);
    // hipMemcpy(hr, dr, IMG_SIZE, hipMemcpyDeviceToHost);
    // hipMemcpy(hg, dg, IMG_SIZE, hipMemcpyDeviceToHost);
    // hipMemcpy(hb, db, IMG_SIZE, hipMemcpyDeviceToHost);
    // for(ui32 y = 0U; y < height; ++y){
    //     for(ui32 x = 0U; x < 3 * width; ++x){
    //         int idx = ((y * width) + x) * 3;
    //         img[idx + 0] = hr[y * width + x];
    //         img[idx + 1] = hg[y * width + x];
    //         img[idx + 2] = hb[y * width + x];
    //     }
    // }

    bits = (BYTE*)FreeImage_GetBits(bitmap);
    for(int y =0; y<height; y++)
    {
        BYTE *pixel = (BYTE*)bits;
        for(int x = 0; x<width; ++x)
        {
            RGBQUAD newcolor;
            int idx = ((y * width) + x) * 3;
            newcolor.rgbRed = img[idx + 0];
            newcolor.rgbGreen = img[idx + 1];
            newcolor.rgbBlue = img[idx + 2];

            if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
                {fprintf(stderr, "(%d, %d) Fail...\n", x, y); }
        pixel+=3;
        }
        // next line
        bits += pitch;
    }

    if(FreeImage_Save (FIF_PNG, bitmap , PathDest , 0))
        cout << "Image successfully saved ! " << endl ;
    FreeImage_DeInitialise(); //Cleanup !

    free(img);
    return 0;
}
