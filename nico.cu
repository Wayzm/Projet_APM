#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <stdlib.h>
#include <cstdio>
#include <cstdlib>
#include <string.h>
#include "FreeImage.h"

#define WIDTH 1920 // I genuinely don't know why these values exist
#define HEIGTH 1024
#define BPP 24
#define ui32 unsigned int
#define nb_threads 32

using namespace std;

//commentaire
__device__ int get_id(void)
{
  int thread_per_block = blockDim.x * blockDim.y * blockDim.z;

  int blockid = gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

  int threadid = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;

  int id = blockid * thread_per_block + threadid;

  return id;
}

__global__ void saturation_r(ui32* d_img, size_t size)
{
    int id = get_id();

    if(id < size)
    {
        d_img[id*3+0] = 255;
    }
}

__global__ void saturation_g(ui32* d_img, size_t size)
{
    int id = get_id();

    if(id < size)
    {
	d_img[id*3+1] = 255;
    }
}

__global__ void saturation_b(ui32* d_img, size_t size)
{
    int id = get_id();

    if(id < size)
    {
        d_img[id*3+2] = 255;
    }
}

__global__ void flou(ui32* d_img, size_t size, size_t width)
{
   int id = get_id();
   ui32 img0, img1, img2;
   if(id < size)
   {
      img0 = d_img[id*3+0];
      img1 = d_img[id*3+1];
      img2 = d_img[id*3+2];

      if(id+1 < size)
      {
         img0 += d_img[id*3+0+3];
         img1 += d_img[id*3+1+3];
         img2 += d_img[id*3+2+3];
      }
      if(id-1 < size && id-1 > 0)
      {
         img0 += d_img[id*3+0-3];
         img1 += d_img[id*3+1-3];
         img2 += d_img[id*3+2-3];
      }
      if(id+width < size)
      {
         img0 += d_img[(id+width)*3+0];
         img1 += d_img[(id+width)*3+1];
         img2 += d_img[(id+width)*3+2];
      }
      if(id-width < size && id-width > 0)
      {
         img0 += d_img[(id-width)*3+0];
         img1 += d_img[(id-width)*3+1];
         img2 += d_img[(id-width)*3+2];
      }
   }

   img0 /= 5;
   img1 /= 5;
   img2 /= 5;

   d_img[id*3+0] = img0;
   d_img[id*3+1] = img1;
   d_img[id*3+2] = img2;
}

__global__ void horizontal_sym(ui32* d_img, ui32* d_tmp, ui32 width, ui32 height){

    // Compute thread id
    const ui32 x = threadIdx.x + blockDim.x * blockIdx.x;
    const ui32 y = threadIdx.y + blockDim.y * blockIdx.y;
    const ui32 idx = y * (3 * width) + x;
    // Compute target destination
    const ui32 xT = x;
    const ui32 yT = height - y;
    const ui32 idxT = yT * 3 * width + xT;
    // Flipping the image
    d_tmp[idxT] = d_img[idx];
}

__global__ void grey_img(ui32* d_img, ui32 width, ui32 height){
    
    int id = get_id();
    // Compute thread id
    // Compute greyed pixel
    if(id < width*height)
    {
       float val  = d_img[id*3+0] * 0.299 + d_img[id*3+1] * 0.587 + d_img[id*3+2] * 0.114;
       // Copy back the greyed pixel
       d_img[id*3+0] = val;
       d_img[id*3+1] = val;
       d_img[id*3+2] = val;
   }
}

__global__ void pop_art()
{

}

int main(int argc, char** argv){

    FreeImage_Initialise();
    const char* PathName="img.jpg";
    const char* PathDest="new_img.png";

    ui32* d_img;
    ui32* d_tmp;
    ui32* dr;
    ui32* dg;
    ui32* db;

    // load and decode a regular file
    FREE_IMAGE_FORMAT fif = FreeImage_GetFileType(PathName);
    if(fif == FIF_UNKNOWN){
        perror("The image either has no signature or the recognition is failing.\n");
        exit(1);
    }
    FIBITMAP* bitmap = FreeImage_Load(fif, PathName, 0);
    if(!bitmap){
        perror("Failed image memory allocation.\n");
        exit(1);
    }

    const ui32 width = FreeImage_GetWidth(bitmap);
    const ui32 height = FreeImage_GetHeight(bitmap);
    const ui32 pitch = FreeImage_GetPitch(bitmap);
    const ui32 IMG_SIZE = sizeof(ui32) * width * height;
    fprintf(stderr, "Processing Image of size %d x %d\n", width, height);

    // Array of IMG
    ui32* img = (ui32*)malloc(3 * IMG_SIZE);
    if(img == NULL){
        perror("Memory allocation for img array failed.\n");
        exit(1);
    }

    ui32* hr = (ui32*)malloc(IMG_SIZE);
    if(hr == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    ui32* hg = (ui32*)malloc(IMG_SIZE);
    if(hg == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    ui32* hb = (ui32*)malloc(IMG_SIZE);
    if(hb == NULL){
        perror("Memory allocation for temporary array failed.\n");
        exit(1);
    }
    // RED, BLUE and GREEN pixels of IMG on device
    hipMalloc((void**)&d_img, 3 * IMG_SIZE);
    hipMalloc((void**)&d_tmp, 3 * IMG_SIZE);
    hipMalloc((void**)&dr, IMG_SIZE);
    hipMalloc((void**)&dg, IMG_SIZE);
    hipMalloc((void**)&db, IMG_SIZE);

   BYTE *bits = (BYTE*)FreeImage_GetBits(bitmap);
    for (ui32 y = 0U; y < height; ++y){
      BYTE *pixel = (BYTE*)bits;
      for (ui32 x = 0U; x < width; ++x){
        int idx = ((y * width) + x) * 3;
        img[idx + 0] = pixel[FI_RGBA_RED];
        img[idx + 1] = pixel[FI_RGBA_GREEN];
        img[idx + 2] = pixel[FI_RGBA_BLUE];
        pixel += 3;
      }
      // next line
      bits += pitch;
    }

    for(ui32 y = 0U; y < height; ++y){
        for(ui32 x = 0U; x < 3 * width; ++x){
            int idx = ((y * width) + x) * 3;
            hr[y * width + x] = img[idx + 0];
            hg[y * width + x] = img[idx + 1];
            hb[y * width + x] = img[idx + 2];
        }
    }
   
    //hipMemcpy(dr, hr, IMG_SIZE, hipMemcpyHostToDevice);
    //hipMemcpy(dg, hg, IMG_SIZE, hipMemcpyHostToDevice);
    //hipMemcpy(db, hb, IMG_SIZE, hipMemcpyHostToDevice);

    hipError_t err = hipMemcpy(d_img,img,3*IMG_SIZE,hipMemcpyHostToDevice);
    if(err != hipSuccess)
       printf("probleme dans hipMemcpy entre\n");

    for(int i = 1; i < argc; i++)
    {

    if(!strcmp(argv[i],"saturation_r"))
    {
       printf("on est ici\n");
       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(width/32+1, height/32+1);   

       saturation_r<<<Num_Blocks,Threads_Per_Blocks>>>(d_img,width*height);
       hipMemcpy(img,d_img,3*IMG_SIZE,hipMemcpyDeviceToHost);
   }

   if(!strcmp(argv[i],"saturation_g"))
   {
       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(3 * width/Threads_Per_Blocks.x, height/Threads_Per_Blocks.y);    

       saturation_g<<<Num_Blocks,Threads_Per_Blocks>>>(d_img,width*height);
       hipMemcpy(img,d_img,3*IMG_SIZE,hipMemcpyDeviceToHost);
   }

   if(!strcmp(argv[i],"saturation_b"))
   {
       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(3 * width/Threads_Per_Blocks.x, height/Threads_Per_Blocks.y);    

       saturation_b<<<Num_Blocks,Threads_Per_Blocks>>>(d_img,width*height);
       hipMemcpy(img,d_img,3*IMG_SIZE,hipMemcpyDeviceToHost);
   }

   if(!strcmp(argv[i],"grey_img"))
   {
       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(3 * width/Threads_Per_Blocks.x, height/Threads_Per_Blocks.y);
       grey_img<<<Num_Blocks,Threads_Per_Blocks>>>(d_img, width, height);
       hipMemcpy(img,d_img,3*IMG_SIZE,hipMemcpyDeviceToHost);
   }

   if(!strcmp(argv[i],"flou"))
   {
       printf("on est dans le flou\n");
       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(3 * width/Threads_Per_Blocks.x, height/Threads_Per_Blocks.y); 
       
       for(int k = 0; k < 100; k++)
       {
          flou<<<Num_Blocks,Threads_Per_Blocks>>>(d_img,width*height,width);
       }
       hipMemcpy(img,d_img,3*IMG_SIZE,hipMemcpyDeviceToHost);
   }  

   if(!strcmp(argv[i],"sym"))
   {
       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(3 * width/Threads_Per_Blocks.x, height/Threads_Per_Blocks.y);
       horizontal_sym<<<Num_Blocks,Threads_Per_Blocks>>>(d_img,d_tmp,width,height);
       hipMemcpy(d_img,d_tmp,3*IMG_SIZE,hipMemcpyDeviceToDevice);
       hipMemcpy(img,d_img,3*IMG_SIZE,hipMemcpyDeviceToHost);
   }

   if(!strcmp(argv[i],"pop_art"))
   {
       hipStream_t stream[5];
       for(int s = 1; s < 5; s++)
       {
          hipStreamCreate(&stream[s]);
       }
       FIBITMAP *split = FreeImage_Rescale(bitmap,width/2,height/2,FILTER_BOX);

       ui32 spitch = GreeImage_GetPitch(split);
       ui32 swidth = GreeImage_GetWidth(split);
       ui32 sheight = GreeImage_GetHeight(split);
       /*ui32* small = (ui32*)malloc(3*sizeof(ui32)*(width/2)*(height/2));
       ui32* bl = (ui32*)malloc(3*sizeof(ui32)*(width/2)*(height/2));
       ui32* br = (ui32*)malloc(3*sizeof(ui32)*(width/2)*(height/2));
       ui32* tl = (ui32*)malloc(3*sizeof(ui32)*(width/2)*(height/2));
       ui32* tr = (ui32*)malloc(3*sizeof(ui32)*(width/2)*(height/2));*/
       ui32* small;
       hipHostMalloc((void**)&small,3*sizeof(ui32)*(width/2)*(height/2));
       ui32* bl;
       hipHostMalloc((void**)&bl,3*sizeof(ui32)*(width/2)*(height/2));
       ui32* br;
       hipHostMalloc((void**)&br,3*sizeof(ui32)*(width/2)*(height/2));
       ui32* tl;
       hipHostMalloc((void**)&tl,3*sizeof(ui32)*(width/2)*(height/2));
       ui32* tr;
       hipHostMalloc((void**)&tr,3*sizeof(ui32)*(width/2)*(height/2));

       BYTE *bits = (BYTE*)FreeImage_GetBits(split);
       for (ui32 y = 0U; y < sheight; ++y){
          BYTE *pixel = (BYTE*)bits;
          for (ui32 x = 0U; x < swidth; ++x){
             int idx = ((y * (swidth)) + x) * 3;
             small[idx + 0] = pixel[FI_RGBA_RED];
             small[idx + 1] = pixel[FI_RGBA_GREEN];
             small[idx + 2] = pixel[FI_RGBA_BLUE];
             pixel += 3;
          }
          // next line
          bits += spitch;
       }
       
       ui32* dbl;
       ui32* dbr;
       ui32* dtl;
       ui32* dtr;
       
       hipMalloc((void**)&dbl, 3*sizeof(ui32)*swidth*sheight);
       hipMalloc((void**)&dbr, 3*sizeof(ui32)*swidth*sheight);
       hipMalloc((void**)&dtl, 3*sizeof(ui32)*swidth*sheight);
       hipMalloc((void**)&dtr, 3*sizeof(ui32)*swidth*sheight);

       dim3 Threads_Per_Blocks(32, 32);
       dim3 Num_Blocks(swidth/32+1, sheight/32+1);

       hipMemcpyAsync(dbl,small,3*sizeof(ui32)*swidth*sheight,hipMemcpyHostToDevice,stream[1]);
       saturation_r<<<Num_Blocks,Threads_Per_Blocks,0,stream[1]>>>(dbl,swidth,sheight);
       hipMemcpyAsync(bl,dbl,3*sizeof(ui32)*swidth*sheight,hipMemcpyDeviceToHost,stream[1]);

       hipMemcpyAsync(dbr,small,3*sizeof(ui32)*swidth*sheight,hipMemcpyHostToDevice,stream[2]);
       saturation_b<<<Num_Blocks,Threads_Per_Blocks,0,stream[2]>>>(dbr,swidth*sheight);
       hipMemcpyAsync(br,dbr,3*sizeof(ui32)*swidth*sheight,hipMemcpyDeviceToHost,stream[2]);

       hipMemcpyAsync(dtl,small,3*sizeof(ui32)*swidth*sheight,hipMemcpyHostToDevice,stream[3]);
       saturation_g<<<Num_Blocks,Threads_Per_Blocks,0,stream[3]>>>(dtl,swidth*sheight);
       hipMemcpyAsync(tl,dtl,3*sizeof(ui32)*swidth*sheight,hipMemcpyDeviceToHost,stream[3]);

       hipMemcpyAsync(dtr,small,3*sizeof(ui32)*swidth*sheight,hipMemcpyHostToDevice,stream[4]);
       grey_img<<<Num_Blocks,Threads_Per_Blocks,0,stream[4]>>>(dtr,swidth,sheight);
       hipMemcpyAsync(tr,dtr,3*sizeof(ui32)*swidth*sheight,hipMemcpyDeviceToHost,stream[4]);
    

       hipDeviceSynchronize();

       for(int j = 0; j < width/2; j++)
       {
           for(int k = 0; k < height/2; k++)
           {
              img[(k*width+j)*3+0] = bl[(k*width/2+j)*3+0];
              img[(k*width+j)*3+1] = bl[(k*width/2+j)*3+1];
              img[(k*width+j)*3+2] = bl[(k*width/2+j)*3+2];
           } 
       }

       for(int j = 0; j < width/2; j++)
       {
           for(int k = 0; k < height/2; k++)
           {
              img[(j+width/2+k*width)*3+0] = br[(k*width/2+j)*3+0];
              img[(j+width/2+k*width)*3+1] = br[(k*width/2+j)*3+1];
              img[(j+width/2+k*width)*3+2] = br[(k*width/2+j)*3+2];
           }
       }

       for(int j = 0; j < width/2; j++)
       {
           for(int k = 0; k < height/2; k++)
           {
              img[(j+(k+height/2)*width)*3+0] = tl[(k*width/2+j)*3+0];
              img[(j+(k+height/2)*width)*3+1] = tl[(k*width/2+j)*3+1];
              img[(j+(k+height/2)*width)*3+2] = tl[(k*width/2+j)*3+2];
           }
       }

       for(int j = 0; j < width/2; j++)
       {
           for(int k = 0; k < height/2; k++)
           {
              img[(j+width/2+(k+height/2)*width)*3+0] = tr[(k*width/2+j)*3+0];
              img[(j+width/2+(k+height/2)*width)*3+1] = tr[(k*width/2+j)*3+1];
              img[(j+width/2+(k+height/2)*width)*3+2] = tr[(k*width/2+j)*3+2];
           }
       }

    }

  }

   err = hipGetLastError();
   if(err != hipSuccess)
        printf("probleme dans hipMemcpy sortie: %s\n",hipGetErrorString(err));

   //err = hipMemcpy(img,d_tmp,3*IMG_SIZE,hipMemcpyDeviceToHost);
   //if(err != hipSuccess)
       // printf("probleme dans hipMemcpy sortie\n");

    bits = (BYTE*)FreeImage_GetBits(bitmap);
    for(int y =0; y<height; y++)
    {
        BYTE *pixel = (BYTE*)bits;
        for(int x = 0; x<width; ++x)
        {
            RGBQUAD newcolor;
            int idx = ((y * width) + x) * 3;
            newcolor.rgbRed = img[idx + 0];
            newcolor.rgbGreen = img[idx + 1];
            newcolor.rgbBlue = img[idx + 2];

            if(!FreeImage_SetPixelColor(bitmap, x, y, &newcolor))
                {fprintf(stderr, "(%d, %d) Fail...\n", x, y); }
        pixel+=3;
        }
        // next line
        bits += pitch;
    }

  if( FreeImage_Save (FIF_PNG, bitmap , PathDest , 0 ))
    cout << "Image successfully saved ! " << endl ;
  FreeImage_DeInitialise(); //Cleanup !
   
   // free(img);
   // free(h_img);
    return 0;
}
